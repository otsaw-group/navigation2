#include "hip/hip_runtime.h"
// Copyright (c) 2024 OTSAW

#include "nav2_mppi_controller/cuda/otsaw_utils.cuh"

// "nav2_costmap_2d/cost_values.hpp"
static constexpr unsigned char NO_INFORMATION = 255;
static constexpr unsigned char LETHAL_OBSTACLE = 254;
// static constexpr unsigned char INSCRIBED_INFLATED_OBSTACLE = 253;
// static constexpr unsigned char MAX_NON_OBSTACLE = 252;
// static constexpr unsigned char FREE_SPACE = 0;

// costmap_2d.cpp
__device__ bool worldToMap(
    // Input
    double wx,
    double wy,
    // Costmap
    unsigned int costmap_size_x,
    unsigned int costmap_size_y,
    double costmap_resolution,
    double costmap_origin_x,
    double costmap_origin_y,
    // Output
    unsigned int & mx,
    unsigned int & my
) {
    if (wx < costmap_origin_x || wy < costmap_origin_y) {
        return false;
    }

    mx = static_cast<unsigned int>((wx - costmap_origin_x) / costmap_resolution);
    my = static_cast<unsigned int>((wy - costmap_origin_y) / costmap_resolution);

    if (mx < costmap_size_x && my < costmap_size_y) {
        return true;
    }
    return false;
}

// footprint_collision_checker.cpp
__device__ double pointCost(
    // Input(0): xy pixel index
    unsigned int x_i,
    unsigned int y_i,
    // Input(1): Costmap
    unsigned char *costmap_arr,
    unsigned int costmap_size_x
) {
    unsigned int index = y_i * costmap_size_x + x_i;
    return costmap_arr[index];
}

// footprint_collision_checker.cpp
__device__ double lineCost(
    // Input(0): 2 points
    int x0, int x1, int y0, int y1,
    // Input(1): Costmap
    unsigned char *costmap_arr,
    unsigned int costmap_size_x
) {
    double line_cost = 0.0;
    double point_cost = -1.0;

    for (LineIterator line(x0, y0, x1, y1); line.isValid(); line.advance()) {
        point_cost = pointCost(
            line.getX(), line.getY(),
            costmap_arr, costmap_size_x
        );   // Score the current point

        // if in collision, no need to continue
        if (point_cost == static_cast<double>(LETHAL_OBSTACLE)) {
            return point_cost;
        }

        if (line_cost < point_cost) {
            line_cost = point_cost;
        }
    }

    return line_cost;
}

// footprint_collision_checker.cpp
__device__ double footprintCostAtPose(
    // Input(0): Pose
    double x, double y, double theta,
    // Input(1): Costmap
    unsigned char *costmap_arr,
    unsigned int costmap_size_x,
    unsigned int costmap_size_y,
    double costmap_resolution,
    double costmap_origin_x,
    double costmap_origin_y,
    // Input(2): footprint
    double *footprint_x,
    double *footprint_y,
    unsigned int footprint_size
) {
    double cos_th = cosf(theta);
    double sin_th = sinf(theta);

    double oriented_footprint_x[10];
    double oriented_footprint_y[10];

    for (unsigned int i = 0; i < footprint_size; ++i) {
        double new_x = x + (footprint_x[i] * cos_th - footprint_y[i] * sin_th);
        double new_y = y + (footprint_x[i] * sin_th + footprint_y[i] * cos_th);
        oriented_footprint_x[i] = new_x;
        oriented_footprint_y[i] = new_y;
    }

    // now we really have to lay down the footprint in the costmap_ grid
    unsigned int x0, x1, y0, y1;
    double footprint_cost = 0.0;

    // get the cell coord of the first point
    if (!worldToMap(oriented_footprint_x[0], oriented_footprint_y[0],
        costmap_size_x, costmap_size_y, costmap_resolution,
        costmap_origin_x, costmap_origin_y,
        x0, y0)
    ) {
        return static_cast<double>(LETHAL_OBSTACLE);
    }

    // cache the start to eliminate a worldToMap call
    unsigned int xstart = x0;
    unsigned int ystart = y0;


    // we need to rasterize each line in the footprint
    for (unsigned int i = 0; i < footprint_size - 1; ++i) {
        // get the cell coord of the second point
        if (!worldToMap(oriented_footprint_x[i + 1], oriented_footprint_y[i + 1],
            costmap_size_x, costmap_size_y, costmap_resolution,
            costmap_origin_x, costmap_origin_y,
            x1, y1)
        ) {
            return static_cast<double>(LETHAL_OBSTACLE);
        }

        footprint_cost = fmaxf(
            lineCost(x0, x1, y0, y1, costmap_arr, costmap_size_x),
            footprint_cost);

        // the second point is next iteration's first point
        x0 = x1;
        y0 = y1;

        // if in collision, no need to continue
        if (footprint_cost == static_cast<double>(LETHAL_OBSTACLE)) {
            return footprint_cost;
        }
    }

    // we also need to connect the first point in the footprint to the last point
    // the last iteration's x1, y1 are the last footprint point's coordinates
    return fmaxf(
        lineCost(xstart, x1, ystart, y1, costmap_arr, costmap_size_x),
        footprint_cost);
}

__device__ void costAtPose(
    // Input(0): Pose
    float x,
    float y,
    float theta,
    // Input(1): Costmap
    unsigned char *costmap_arr,
    unsigned int costmap_size_x,
    unsigned int costmap_size_y,
    double costmap_resolution,
    double costmap_origin_x,
    double costmap_origin_y,
    // Input(2): Footprint
    double *footprint_x,
    double *footprint_y,
    unsigned int footprint_size,
    // Input(3): Config
    bool consider_footprint,
    float possibly_inscribed_cost,
    // Output
    float &cost,
    bool &using_footprint,
    // others
    bool debug = false
) {
    using_footprint = false;
    unsigned int x_i, y_i;

    if (!worldToMap(x, y,
        costmap_size_x, costmap_size_y, costmap_resolution,
        costmap_origin_x, costmap_origin_y,
        x_i, y_i)
    ) {
        cost = NO_INFORMATION;
        return;
    }

    cost = pointCost(x_i, y_i, costmap_arr, costmap_size_x);
    if (debug) {
        printf("[GPU] pointCost(%d, %d)=%.2f\n",
            x_i, y_i, cost);
    }

    if (consider_footprint &&
        (cost >= possibly_inscribed_cost || possibly_inscribed_cost < 1.0f))
    {
        // printf("footprintCostAtPose(%.2f, %.2f, %.2f)", x, y, theta);
        cost = footprintCostAtPose(
            // Input(0): Pose
            x, y, theta,
            // Input(1): Costmap
            costmap_arr,
            costmap_size_x,
            costmap_size_y,
            costmap_resolution,
            costmap_origin_x,
            costmap_origin_y,
            // Input(2): footprint
            footprint_x,
            footprint_y,
            footprint_size
        );
        if (debug) {
            printf("[GPU] footprintCostAtPose=%.2f\n", cost);
        }
        using_footprint = true;
    }
}

__global__ void poseCostKernel(
    // Input(0): Trajectories
    float *traj_x,
    float *traj_y,
    float *traj_yaws,
    unsigned int batch_size,
    unsigned int time_steps,
    // Input(1): Costmap
    unsigned char *costmap_arr,
    unsigned int costmap_size_x,
    unsigned int costmap_size_y,
    double costmap_resolution,
    double costmap_origin_x,
    double costmap_origin_y,
    // Input(2): Footprint
    double *footprint_x,
    double *footprint_y,
    unsigned int footprint_size,
    // Input(3): Config
    bool consider_footprint,
    float possibly_inscribed_cost,
    // Output
    float *out_pose_cost,
    bool *out_using_footprint,
    // Iteration limit
    int n
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        // Process
        float cost;
        bool using_footprint;

        costAtPose(
            // Input(0): Pose
            traj_x[tid],
            traj_y[tid],
            traj_yaws[tid],
            // Input(1): Costmap
            costmap_arr,
            costmap_size_x,
            costmap_size_y,
            costmap_resolution,
            costmap_origin_x,
            costmap_origin_y,
            // Input(2): Footprint
            footprint_x,
            footprint_y,
            footprint_size,
            // Input(3): Config
            consider_footprint,
            possibly_inscribed_cost,
            // Output
            cost,
            using_footprint,
            // Debug (i*time_steps+j)
            (tid == 0*time_steps+55)
        );
        
        // batchsize 2000 x timestep 56
        out_pose_cost[tid] = cost;
        out_using_footprint[tid] = using_footprint;

        if (tid == 0*time_steps+55) {
            unsigned int i = tid / time_steps;
            unsigned int j = tid - (i * time_steps);
            
            printf("[GPU] tid=%d, i=%d, j=%d, x=%f, y=%f, yaws=%f, pose_cost=%.2f, using_footprint=%d\n",
                tid,
                i,
                j,
                traj_x[tid],
                traj_y[tid],
                traj_yaws[tid],
                out_pose_cost[tid],
                out_using_footprint[tid] ? 1 : 0
            );
        }
    }
}

void calc_cost_at_pose(
    // Input(0): Trajectories
    std::vector<float> traj_x,
    std::vector<float> traj_y,
    std::vector<float> traj_yaws,
    unsigned int batch_size,
    unsigned int time_steps,
    // Input(1): Costmap
    unsigned char * costmap_arr,
    unsigned int costmap_size_x,
    unsigned int costmap_size_y,
    double costmap_resolution,
    double costmap_origin_x,
    double costmap_origin_y,
    // Input(2): Footprint
    std::vector<double> footprint_x,
    std::vector<double> footprint_y,
    unsigned int footprint_size,
    // Input(3): Config
    bool consider_footprint,
    float possibly_inscribed_cost,
    // Output:
    std::vector<float>& out_pose_cost,
    std::vector<bool>& out_using_footprint
) {
    // Timing
    // hipEvent_t t0, t_end;
    // hipEventCreate(&t0);
    // hipEventCreate(&t_end);
    // hipEventRecord(t0);

    // Max number of triggers
    int N = traj_x.size();

    thrust::device_vector<unsigned char> d_vec_costmap_arr(costmap_arr,
        costmap_arr + costmap_size_x * costmap_size_y);
    unsigned char* d_costmap_arr = thrust::raw_pointer_cast(d_vec_costmap_arr.data());

    thrust::device_vector<float> d_vec_traj_x = traj_x;
    float* d_traj_x = thrust::raw_pointer_cast(d_vec_traj_x.data());

    thrust::device_vector<float> d_vec_traj_y = traj_y;
    float* d_traj_y = thrust::raw_pointer_cast(d_vec_traj_y.data());

    thrust::device_vector<float> d_vec_traj_yaws = traj_yaws;
    float* d_traj_yaws = thrust::raw_pointer_cast(d_vec_traj_yaws.data());

    thrust::device_vector<double> d_vec_footprint_x = footprint_x;
    double* d_footprint_x = thrust::raw_pointer_cast(d_vec_footprint_x.data());

    thrust::device_vector<double> d_vec_footprint_y = footprint_y;
    double* d_footprint_y = thrust::raw_pointer_cast(d_vec_footprint_y.data());

    thrust::device_vector<float> d_vec_pose_cost(batch_size);
    float* d_pose_cost = thrust::raw_pointer_cast(d_vec_pose_cost.data());

    thrust::device_vector<bool> d_vec_using_footprint(batch_size);
    bool* d_using_footprint = thrust::raw_pointer_cast(d_vec_using_footprint.data());

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    poseCostKernel<<<numBlocks, blockSize>>>(
        // Input(0): Trajectories
        d_traj_x,
        d_traj_y,
        d_traj_yaws,
        batch_size,
        time_steps,
        // Input(1): Costmap
        d_costmap_arr,
        costmap_size_x,
        costmap_size_y,
        costmap_resolution,
        costmap_origin_x,
        costmap_origin_y,
        // Input(2): Footprint
        d_footprint_x,
        d_footprint_y,
        footprint_size,
        // Input(3): Config
        consider_footprint,
        possibly_inscribed_cost,
        // Output
        d_pose_cost,
        d_using_footprint,
        // Iteration limit
        N
    );

    hipDeviceSynchronize();

    // Output:
    thrust::host_vector<float> h_pose_cost = d_vec_pose_cost;
    out_pose_cost = std::vector<float>(h_pose_cost.begin(), h_pose_cost.end());

    thrust::host_vector<bool> h_using_footprint = d_vec_using_footprint;
    out_using_footprint = std::vector<bool>(h_using_footprint.begin(), h_using_footprint.end());

    // Calculate elapsed time
    // hipEventRecord(t_end);
    // hipEventSynchronize(t_end);
    // float milliseconds = 0;
    // hipEventElapsedTime(&milliseconds, t0, t_end);
    // std::cout << "t_end-t0: " << milliseconds << " ms" << std::endl;
}