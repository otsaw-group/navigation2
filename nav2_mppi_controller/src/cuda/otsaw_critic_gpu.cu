#include "hip/hip_runtime.h"
// Copyright (c) 2024 OTSAW

#include "nav2_mppi_controller/cuda/otsaw_critic_gpu.cuh"

__global__ void constraintKernel(
    float *out,
    float *vx,
    float *vy,
    float max_vel,
    float min_vel,
    float dt,
    int n
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        float sgn = (vx[tid] > 0.0) ? 1.0 : -1.0;
        float vel_total = sgn * sqrtf(vx[tid]*vx[tid] + vy[tid]*vy[tid]);
        
        float total_error = 0.0;
        if (vel_total > max_vel) total_error += (vel_total - max_vel);
        if (vel_total < min_vel) total_error += (min_vel - vel_total);
        
        // TODO: support ackerman model

        // Calculate cost output
        out[tid] = (total_error) * dt;

        // printf("tid=%d, vx=%.2f, range=(%.2f, %.2f), vel_total=%.2f, total_error=%.2f, out=%.2f\n",
        //     tid, vx[tid], 
        //     min_vel, max_vel,
        //     vel_total,
        //     total_error,
        //     out[tid]);
    }
}

std::vector<float> calc_constraint_critics_cost(
    std::vector<float> vx,
    std::vector<float> vy,
    float max_vel,
    float min_vel,
    float dt
) {
    int N = vx.size();
    thrust::device_vector<float> d_vx = vx;
    thrust::device_vector<float> d_vy = vy;
    thrust::device_vector<float> d_out(N);

    // Extract raw pointersvec_vx
    float* raw_ptr_vx = thrust::raw_pointer_cast(d_vx.data());
    float* raw_ptr_vy = thrust::raw_pointer_cast(d_vy.data());
    float* raw_ptr_out = thrust::raw_pointer_cast(d_out.data());

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    constraintKernel<<<numBlocks, blockSize>>>(
        raw_ptr_out,
        raw_ptr_vx,
        raw_ptr_vy,
        max_vel,
        min_vel,
        dt,
        N
    );

    hipDeviceSynchronize();

    thrust::host_vector<float> h_out = d_out;
    std::vector<float> out(h_out.begin(), h_out.end());

    return out;
}

__global__ void obstacleKernel(
    unsigned char *costmap_arr,
    unsigned int costmap_size_x,
    unsigned int costmap_size_y,
    double costmap_resolution,
    double costmap_origin_x,
    double costmap_origin_y,
    float *traj_x,
    float *traj_y,
    float *traj_yaws,
    unsigned int batch_size,
    unsigned int time_steps,
    float *raw_cost,
    float *repulsive_cost,
    int n
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        unsigned int j = tid / batch_size;  // pose in traj index
        unsigned int i = tid - (j * batch_size);  // traj in batch of trajectories

        // Process
        printf("tid=%d, i=%d, j=%d, x=%.2f, y=%.2f, yaws=%.2f\n",
            tid,
            i,
            j,
            traj_x[tid],
            traj_y[tid],
            traj_yaws[tid]
        );
    }
}

void calc_obstacle_critics_cost(
    unsigned char * costmap_arr,
    unsigned int costmap_size_x,
    unsigned int costmap_size_y,
    double costmap_resolution,
    double costmap_origin_x,
    double costmap_origin_y,
    std::vector<float> traj_x,
    std::vector<float> traj_y,
    std::vector<float> traj_yaws,
    unsigned int batch_size,
    unsigned int time_steps,
    std::vector<float>& raw_cost,
    std::vector<float>& repulsive_cost
) {
    // Max number of triggers
    int N = traj_x.size();
    // printf("traj_x.size()=%d, batch_size=%d, timestep=%d",
    //     traj_x.size(), batch_size, time_steps);

    thrust::device_vector<unsigned char> d_costmap_arr(costmap_arr,
        costmap_arr + costmap_size_x * costmap_size_y);
    thrust::device_vector<float> d_traj_x = traj_x;
    thrust::device_vector<float> d_traj_y = traj_y;
    thrust::device_vector<float> d_traj_yaws = traj_yaws;
    thrust::device_vector<float> d_raw_cost = raw_cost;
    thrust::device_vector<float> d_repulsive_cost = repulsive_cost;

    unsigned char* ptr_costmap_arr = thrust::raw_pointer_cast(d_costmap_arr.data());
    float* ptr_traj_x = thrust::raw_pointer_cast(d_traj_x.data());
    float* ptr_traj_y = thrust::raw_pointer_cast(d_traj_y.data());
    float* ptr_traj_yaws = thrust::raw_pointer_cast(d_traj_yaws.data());
    float* ptr_raw_cost = thrust::raw_pointer_cast(d_raw_cost.data());
    float* ptr_repulsive_cost = thrust::raw_pointer_cast(d_repulsive_cost.data());

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    obstacleKernel<<<numBlocks, blockSize>>>(
        ptr_costmap_arr,
        costmap_size_x,
        costmap_size_y,
        costmap_resolution,
        costmap_origin_x,
        costmap_origin_y,
        ptr_traj_x,
        ptr_traj_y,
        ptr_traj_yaws,
        batch_size,
        time_steps,
        ptr_raw_cost,
        ptr_repulsive_cost,
        N
    );

    hipDeviceSynchronize();

    // Output
    thrust::host_vector<float> h_raw_cost = d_raw_cost;
    raw_cost = std::vector<float>(h_raw_cost.begin(), h_raw_cost.end());

    thrust::host_vector<float> h_repulsive_cost = d_repulsive_cost;
    repulsive_cost = std::vector<float>(h_repulsive_cost.begin(), h_repulsive_cost.end());
}