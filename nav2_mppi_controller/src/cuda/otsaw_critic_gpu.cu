#include "hip/hip_runtime.h"
// Copyright (c) 2024 OTSAW

#include "nav2_mppi_controller/cuda/otsaw_critic_gpu.cuh"

__global__ void constraintKernel(
    float *out,
    float *vx,
    float *vy,
    float max_vel,
    float min_vel,
    float dt,
    int n
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        float sgn = (vx[tid] > 0.0) ? 1.0 : -1.0;
        float vel_total = sgn * sqrtf(vx[tid]*vx[tid] + vy[tid]*vy[tid]);
        
        float total_error = 0.0;
        if (vel_total > max_vel) total_error += (vel_total - max_vel);
        if (vel_total < min_vel) total_error += (min_vel - vel_total);
        
        // TODO: support ackerman model

        // Calculate cost output
        out[tid] = (total_error) * dt;

        // printf("tid=%d, vx=%.2f, range=(%.2f, %.2f), vel_total=%.2f, total_error=%.2f, out=%.2f\n",
        //     tid, vx[tid], 
        //     min_vel, max_vel,
        //     vel_total,
        //     total_error,
        //     out[tid]);
    }
}

std::vector<float> calc_constraint_critics_cost(
    std::vector<float> vx,
    std::vector<float> vy,
    float max_vel,
    float min_vel,
    float dt
) {
    int N = vx.size();
    thrust::device_vector<float> d_vx = vx;
    thrust::device_vector<float> d_vy = vy;
    thrust::device_vector<float> d_out(N);

    // Extract raw pointersvec_vx
    float* raw_ptr_vx = thrust::raw_pointer_cast(d_vx.data());
    float* raw_ptr_vy = thrust::raw_pointer_cast(d_vy.data());
    float* raw_ptr_out = thrust::raw_pointer_cast(d_out.data());

    // Launch kernel
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    constraintKernel<<<numBlocks, blockSize>>>(
        raw_ptr_out,
        raw_ptr_vx,
        raw_ptr_vy,
        max_vel,
        min_vel,
        dt,
        N
    );

    hipDeviceSynchronize();

    thrust::host_vector<float> h_out = d_out;
    std::vector<float> out(h_out.begin(), h_out.end());

    return out;
}
