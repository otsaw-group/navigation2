#include "hip/hip_runtime.h"
// Copyright (c) 2024 OTSAW

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "nav2_mppi_controller/cuda/otsaw_critic_gpu.cuh"

#define N 10000000
#define MAX_ERR 1e-6

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Handling arbitrary vector size
    if (tid < n){
        out[tid] = a[tid] + b[tid];
    }
}

float test_gpu_fn() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device Number: %d\n", i);
      printf("  Device name: %s\n", prop.name);
      printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
      printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
      printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

    float *a, *b, *out;
    float *d_a, *d_b, *d_out; 

    // Allocate host memory
    a   = (float*)malloc(sizeof(float) * N);
    b   = (float*)malloc(sizeof(float) * N);
    out = (float*)malloc(sizeof(float) * N);

    // Initialize host arrays
    for(int i = 0; i < N; i++){
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory 
    gpuErrchk( hipMalloc((void**)&d_a, sizeof(float) * N));
    gpuErrchk( hipMalloc((void**)&d_b, sizeof(float) * N));
    gpuErrchk( hipMalloc((void**)&d_out, sizeof(float) * N));

    // Transfer data from host to device memory
    gpuErrchk( hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice));

    // Executing kernel 
    int block_size = 256;
    int grid_size = ((N + block_size) / block_size);
    vector_add<<<grid_size,block_size>>>(d_out, d_a, d_b, N);

    gpuErrchk( hipDeviceSynchronize() );

    // Transfer data back to host memory
    gpuErrchk( hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost));

    // Verification
    printf("a[0]=%f\n", a[0]);
    printf("b[0]=%f\n", b[0]);
    printf("out[0]=%f\n", out[0]);
    float ret = out[0];

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a); 
    free(b); 
    free(out);

    return ret;
}
